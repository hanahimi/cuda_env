/**
������豸����ʶ
�������ǵĳ����Զ�ͨ������cuda API��������豸��Ŀ������
*/
#include "hip/hip_runtime.h"

#include "hip/driver_types.h"
#include <stdio.h>

int main()
{
	hipError_t cudaStatus;
	int num = 0;
	hipDeviceProp_t prop;
	cudaStatus = hipGetDeviceCount(&num);
	
	// Choose which GPU to run on, change this on a multi-GPU system.
	//cudaStatus = cudaSetDevice(0);
	
	for (int i = 0; i < num; i++)
	{
		hipGetDeviceProperties(&prop, i);
	}

	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;

}
/**
ʹ����cudaSetDevice(0)���������0��ʾ���������ĵ�һ���豸�ţ�
����ж���豸������Ϊ0,1,2...��

�ٿ����Ǳ�����ӵĴ��룬�и�����cudaGetDeviceCount(&num)���������������ȡ�豸������
��������ѡ������CUDA������豸��ȡֵ����0,1,...num-1�����ǿ���һ����ö���豸��
����cudaGetDeviceProperties(&prop)���������,Ȼ������һ������ɸѡ�㷨��
�ҵ����������Ӧ�õ��Ǹ��豸��opt��Ȼ�����cudaSetDevice(opt)����ѡ����豸��
ѡ���׼���ԴӴ����������汾���ơ����Ƶȸ����Ƕȳ�����
*/

