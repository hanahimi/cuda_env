/**
CUDA从入门到精通（六）：块并行
块并行相当于操作系统中多进程的情况，上节说到，CUDA有线程组（线程块）的概念，
将一组线程组织到一起，共同分配一部分资源，然后内部调度执行。
线程块与线程块之间，毫无瓜葛。这有利于做更粗粒度的并行。
我们将上一节的代码改为块并行版本如下：
*/
#include "hip/hip_runtime.h"


#include <stdio.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = blockIdx.x;
    c[i] = a[i] + b[i];
}

int main()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<size, 1>>>(dev_c, dev_a, dev_b);
	/**
	和上一节相比，只有这两行有改变，<<<>>>里第一个参数改成了size，第二个改成了1，
	表示我们分配size个线程块，每个线程块仅包含1个线程，总共还是有5个线程。
	这5个线程相互独立，执行核函数得到相应的结果，
	与上一节不同的是，每个线程获取id的方式变为int i = blockIdx.x；这是线程块ID
	*/
    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}


/**
线程并行和块并行的区别在哪里？

线程并行是细粒度并行，调度效率高；
块并行是粗粒度并行，每次调度都要重新分配资源，
有时资源只有一份，那么所有线程块都只能排成一队，串行执行。

那是不是我们所有时候都应该用线程并行，尽可能不用块并行？

当然不是，我们的任务有时可以采用分治法，将一个大问题分解为几个小规模问题，
将这些小规模问题分别用一个线程块实现
线程块内可以采用细粒度的线程并行，而块之间为粗粒度并行，这样可以充分利用硬件资源，降低线程并行的计算复杂度
适当分解，降低规模，在一些矩阵乘法、向量内积计算应用中可以得到充分的展示。
*/